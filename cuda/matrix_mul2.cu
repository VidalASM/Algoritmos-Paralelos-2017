
#include <hip/hip_runtime.h>
#include <stdio.h>

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *d_M , float *d_N , float *d_P , int Width) {
    //2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0.0;

    for(int k = 0; k < Width ; ++k) {
        float Mdelement = d_M[ty*Width + k];
        float Ndelement = d_N[k*Width + tx];
        Pvalue += (Mdelement*Ndelement);
    }

    d_P[ty*Width + tx] = Pvalue;
}

int main(void) {
    const int Width = 8;
    float h_Ma[Width][Width], h_Mb[Width][Width], h_Mc[Width][Width];
    float *Md, *Nd, *Pd;
    int i, j;
    
    //input in host array
    for (i = 0; i < Width; i++) {
        for (j = 0; j < Width; j++) {
            h_Ma[i][j] = 1 ;
            h_Mb[i][j] = 2 ;
        }
    }

    //MatrixMultiplication(h_Ma, h_Mb, h_Mc, Width);
    int size = (Width*Width)*sizeof(float);

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md, h_Ma, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd, h_Mb, size, hipMemcpyHostToDevice);

    //Allocate P on the device
    hipMalloc((void**)&Pd,size);

    //Setup the execution configuration
    dim3 dimBlock(Width,Width);
    dim3 dimGrid(1,1);

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

    //Transfer P from device to host
    hipMemcpy(h_Mc,Pd,size,hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    
    //printf the result array
    for (i = 0; i < Width; i++) {
        for (j = 0; j < Width; j++) {
            printf("%f   ", h_Mc[i][j]);
        }
        printf("\n");
    }

    return 0;
}
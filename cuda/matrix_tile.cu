
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define TILE_WIDTH 2

__global__
void MatrixMulKernel(float *d_M , float *d_N , float *d_P , int Width) {
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	float Pvalue = 0;
	// Loop over the d_M and d_N tiles required to compute d_P element
	// ph indicate number of phase
	for (int ph = 0; ph < ceil(Width/(float)TILE_WIDTH); ++ph) {
		
		// Collaborative loading of d_M and d_N tiles into shared memory
		if ((Row < Width) && ((ph*TILE_WIDTH + tx) < Width))
			Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
		else Mds[ty][tx] = 0.0;
		if (((ph*TILE_WIDTH + ty) < Width) && (Col < Width))
			Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
		else Nds[ty][tx] = 0.0;
		
		__syncthreads(); // for syncronizeing the threads

		for (int k = 0; k < TILE_WIDTH; ++k) {
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads(); // for syncronizeing the threads
	}
	if ((Row < Width) && (Col < Width))
		d_P[Row*Width + Col] = Pvalue;
}

int main()
{
	const int Width = 6;
	float h_Ma[Width][Width], h_Mb[Width][Width], h_Mc[Width][Width];
	float *d_Ma, *d_Mb, *d_Mc; // device array
	int i, j;

	/*h_Ma = (float**) malloc((Width)*sizeof(float));
	h_Mb = (float**) malloc((Width)*sizeof(float));
	h_Mc = (float**) malloc((Width)*sizeof(float));
	for (i = 0; i < Width; i++) {
		h_Ma[i] = (float*) malloc((Width)*sizeof(float));
		h_Mb[i] = (float*) malloc((Width)*sizeof(float));
		h_Mc[i] = (float*) malloc((Width)*sizeof(float));
	}*/
	
	//input in host array
	for (i = 0; i < Width; i++)	{
		for (j = 0; j < Width; j++) {
			h_Ma[i][j] = 1 ;
			h_Mb[i][j] = 2 ;
		}
	}

	int size = (Width*Width)*sizeof(float);

	//create device array cudaMalloc ( (void **)&array_name, sizeofmatrixinbytes);
	hipMalloc((void **) &d_Ma, size);
	hipMalloc((void **) &d_Mb, size);

	//copy host array to device array; cudaMemcpy ( dest , source , WIDTH , direction )
	hipMemcpy(d_Ma, h_Ma, size, hipMemcpyHostToDevice);
	hipMemcpy(d_Mb, h_Mb, size, hipMemcpyHostToDevice);

	//allocating memory for resultent device array
	hipMalloc((void **) &d_Mc, size);

	//calling kernal
	dim3 dimGrid(Width/TILE_WIDTH, Width/TILE_WIDTH, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	MatrixMulKernel<<<dimGrid,dimBlock>>>(d_Ma, d_Mb, d_Mc, Width);

	// all gpu function blocked till kernel is working
	//copy back result_array_d to result_array_h
	hipMemcpy(h_Mc, d_Mc, size, hipMemcpyDeviceToHost);

	//printf the result array
	for (i = 0; i < Width; i++) {
		for (j = 0; j < Width; j++) {
			printf("%f   ", h_Mc[i][j]);
		}
		printf("\n");
	}

	hipFree(d_Ma);
	hipFree(d_Mb);
	hipFree(d_Mc);
	
	//system("pause") ;
	return 0;
}
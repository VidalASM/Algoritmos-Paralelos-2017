
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>       /* round() */

typedef struct bmpFileHeader
{
  /* 2 bytes de identificación */
  unsigned int size;        /* Tamaño del archivo */
  unsigned short resv1;       /* Reservado */
  unsigned short resv2;       /* Reservado */
  unsigned int offset;      /* Offset hasta hasta los datos de imagen */
} bmpFileHeader;

typedef struct bmpInfoHeader
{
  unsigned int headersize;  /* Tamaño de la cabecera */
  unsigned int width;       /* Ancho */
  unsigned int height;      /* Alto */
  unsigned short planes;      /* Planos de color (Siempre 1) */
  unsigned short bpp;         /* bits por pixel */
  unsigned int compress;    /* compresión */
  unsigned int imgsize;     /* tamaño de los datos de imagen */
  unsigned int bpmx;        /* Resolución X en bits por metro */
  unsigned int bpmy;        /* Resolución Y en bits por metro */
  unsigned int colors;      /* colors used en la paleta */
  unsigned int imxtcolors;  /* Colores importantes. 0 si son todos */
} bmpInfoHeader;

unsigned char *LoadBMP(char *filename, bmpInfoHeader *bInfoHeader);
void DisplayInfo(bmpInfoHeader *info);
void TextDisplay(bmpInfoHeader *info, unsigned char *img);

void SaveBMP(char *filename, bmpInfoHeader *info, unsigned char *imgdata);

__global__
void escalaGrises(unsigned char *img, unsigned char *grayimg, int numRows, int numCols)
{
  int Col = threadIdx.x + blockIdx.x*blockDim.x;
  int Row = threadIdx.y + blockIdx.y*blockDim.y;

  if(Col < numCols && Row < numRows)
  {
    int grayOffset = Col + Row*numCols;
    int rgbOffset = grayOffset*3;
    unsigned char r = img[rgbOffset + 2];
    unsigned char g = img[rgbOffset + 1];
    unsigned char b = img[rgbOffset    ];
    grayimg[grayOffset] = 0.21f*r + 0.71f*g + 0.07*b;
  }
}

int main()
{
  bmpInfoHeader info;
  unsigned char *img, *im2;
  int blockThreadSize, numberOfBlocks;
  int numRows, numCols;

  img = LoadBMP("magia_sonrisa.bmp", &info);
  DisplayInfo(&info);
  TextDisplay(&info, img);
  numRows = info.height;
  numCols = info.width;

  blockThreadSize = 32;
  numberOfBlocks = 1 + ((numRows - 1) / blockThreadSize);

  const int isize = (numRows*numCols)*sizeof(char);
  hipMalloc((void**)&im2, isize);

  const dim3 blockSize(blockThreadSize, blockThreadSize, 1);
  const dim3 gridSize(numberOfBlocks , ceil(numCols/blockThreadSize), 1);
  escalaGrises<<<gridSize, blockSize>>>(img, im2, numRows, numCols);

  hipMemcpy(img, im2, isize, hipMemcpyDeviceToHost);
  hipFree(im2);

  SaveBMP("resultado.bmp", &info, img);

  free(img);

  return 0;
}

void TextDisplay(bmpInfoHeader *info, unsigned char *img)
{
  int x, y;
  /* Reducimos la resolución vertical y horizontal para que la imagen entre en pantalla */
  static const int reduccionX=6, reduccionY=4;
  /* Si la componente supera el umbral, el color se marcará como 1. */
  static const int umbral=90;
  /* Asignamos caracteres a los colores en pantalla */
  static unsigned char colores[9]=" bgfrRGB";
  int r,g,b;

  /* Dibujamos la imagen */
  for (y=info->height; y>0; y-=reduccionY)
  {
    for (x=0; x<info->width; x+=reduccionX)
    {
      b=(img[3*(x+y*info->width)]>umbral);
      g=(img[3*(x+y*info->width)+1]>umbral);
      r=(img[3*(x+y*info->width)+2]>umbral);

      printf("%c", colores[b+g*2+r*4]);
    }
    printf("\n");
  }
}

unsigned char *LoadBMP(char *filename, bmpInfoHeader *bInfoHeader)
{

  FILE *f;
  bmpFileHeader header;     /* cabecera */
  unsigned char *imgdata;   /* datos de imagen */
  unsigned short type;            /* 2 bytes identificativos */

  f=fopen (filename, "r");
  if (!f)
    return NULL;            /* Si no podemos leer, no hay imagen*/

  /* Leemos los dos primeros bytes */
  fread(&type, sizeof(unsigned short), 1, f);
  if (type !=0x4D42)        /* Comprobamos el formato */
  {
    fclose(f);
    return NULL;
  }

  /* Leemos la cabecera de fichero completa */
  fread(&header, sizeof(bmpFileHeader), 1, f);

  /* Leemos la cabecera de información completa */
  fread(bInfoHeader, sizeof(bmpInfoHeader), 1, f);

  /* Reservamos memoria para la imagen, ¿cuánta? 
     Tanto como indique imgsize */
  imgdata=(unsigned char*)malloc(bInfoHeader->imgsize);

  /* Nos situamos en el sitio donde empiezan los datos de imagen,
   nos lo indica el offset de la cabecera de fichero*/
  fseek(f, header.offset, SEEK_SET);

  /* Leemos los datos de imagen, tantos bytes como imgsize */
  fread(imgdata, bInfoHeader->imgsize,1, f);

  /* Cerramos */
  fclose(f);

  /* Devolvemos la imagen */
  return imgdata;
}

void DisplayInfo(bmpInfoHeader *info)
{
  printf("Tamaño de la cabecera: %u\n", info->headersize);
  printf("Anchura: %d\n", info->width);
  printf("Altura: %d\n", info->height);
  printf("Planos (1): %d\n", info->planes);
  printf("Bits por pixel: %d\n", info->bpp);
  printf("Compresión: %d\n", info->compress);
  printf("Tamaño de datos de imagen: %u\n", info->imgsize);
  printf("Resolucón horizontal: %u\n", info->bpmx);
  printf("Resolucón vertical: %u\n", info->bpmy);
  printf("Colores en paleta: %d\n", info->colors);
  printf("Colores importantes: %d\n", info->imxtcolors);
}

void SaveBMP(char *filename, bmpInfoHeader *info, unsigned char *imgdata)
{
  bmpFileHeader header;
  FILE *f;
  unsigned short type;
  
  f=fopen(filename, "w+");
  header.size = info->imgsize + sizeof(bmpFileHeader) + sizeof(bmpInfoHeader);
  /* header.resv1=0; */
  /* header.resv2=1; */
  
  /* El offset será el tamaño de las dos cabeceras + 2 (información de fichero)*/
  header.offset = sizeof(bmpFileHeader) + sizeof(bmpInfoHeader) + 2;
  
  /* Escribimos la identificación del archivo */
  type = 0x4D42;
  fwrite(&type, sizeof(type),1,f);
  
  /* Escribimos la cabecera de fichero */
  fwrite(&header, sizeof(bmpFileHeader),1,f);
  
  /* Escribimos la información básica de la imagen */
  fwrite(info, sizeof(bmpInfoHeader),1,f);
  
  /* Escribimos la imagen */
  fwrite(imgdata, info->imgsize, 1, f);
  
  fclose(f);
}